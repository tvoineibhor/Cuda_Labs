#include "Callback.h"


void MesurePerfomance(void(*f)(float *, float *, int), float * devInputArr, float * devOutputArr, int size, int block_size, char* type)
{
	float time_elapsed = 0;
	hipEvent_t start;
	hipEvent_t stop;

	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));
	CUDA_CHECK_ERROR(hipEventRecord(start, 0));

	for (int i = 0; i < 1000; i++)
	{
		f << <size / block_size, block_size >> > (devInputArr, devOutputArr, size);
	}

	CUDA_CHECK_ERROR(hipEventRecord(stop, 0));
	CUDA_CHECK_ERROR(hipEventSynchronize(stop));
	CUDA_CHECK_ERROR(hipEventElapsedTime(&time_elapsed, start, stop));

	printf("%s done in %f milliseconds\n", type, time_elapsed);
}
