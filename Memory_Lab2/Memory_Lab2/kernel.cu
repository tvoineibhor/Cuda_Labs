#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "Callback.h"


#ifndef __HIPCC__ 
	#define __HIPCC__
#endif

#include <hip/device_functions.h>

#include <stdlib.h>
#include <stdio.h>

#include <iostream>
#include <vector>

using namespace std;

#define N 100000
#define block_size 1000

texture<float, hipTextureType1D, hipReadModeElementType> texX;

__global__ void textureInverse(float * inputArr, float * outputArr, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int id_reverse = (size - 1 - idx);

	if (idx < size)
		outputArr[idx] = tex1Dfetch(texX, float(id_reverse));
}

__global__ void globalInverse(float * inputArr, float * outputArr, int size)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int id_reverse = (blockDim.x * gridDim.x) - 1 - idx;

	if (idx < size)
		outputArr[idx] = inputArr[id_reverse];
}

__global__ void sharedInverse(float * inputArr, float * outputArr, int size)
{

	__shared__ float temp[block_size];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int id_reverse = (blockDim.x * gridDim.x) - 1 - idx;


	temp[threadIdx.x] = inputArr[id_reverse];

	__syncthreads();

	if (idx < size)
		outputArr[idx] = temp[threadIdx.x];

}


int main()
{

	float* inputArr = new float[N];
	float* outputArr = new float[N];

	for (int i = 0; i < N; i++)
	{
		inputArr[i] = i + 1;
	}

	float* devInputArr;
	float* devOutputArr;
	
	CUDA_CHECK_ERROR(hipMalloc((void**)&devInputArr, N * sizeof(float)));
	CUDA_CHECK_ERROR(hipMalloc((void**)&devOutputArr, N * sizeof(float)));

	CUDA_CHECK_ERROR(hipMemcpy(devInputArr, inputArr, N * sizeof(float), hipMemcpyHostToDevice));

	texX.normalized = false;
	CUDA_CHECK_ERROR(hipBindTexture((size_t)0, &texX, devInputArr, &texX.channelDesc, N * sizeof(float)));

	//hipMemcpyToSymbol(HIP_SYMBOL(temp_c), inputArr, N * sizeof(float), 0);

	MesurePerfomance(globalInverse, devInputArr, devOutputArr, N, block_size, "Global");
	//MesurePerfomance(constantInverse, temp_c, devOutputArr, N, block_size, "Constant");
	MesurePerfomance(sharedInverse, devInputArr, devOutputArr, N, block_size, "Shared");
	MesurePerfomance(textureInverse, devInputArr, devOutputArr, N, block_size, "Texture");

	CUDA_CHECK_ERROR(hipMemcpy(outputArr, devOutputArr, N * sizeof(float), hipMemcpyDeviceToHost));

	delete inputArr;
	delete outputArr;

	CUDA_CHECK_ERROR(hipFree(devInputArr));
	CUDA_CHECK_ERROR(hipFree(devOutputArr));

	CUDA_CHECK_ERROR(hipUnbindTexture(&texX));

	cout << endl;
}
