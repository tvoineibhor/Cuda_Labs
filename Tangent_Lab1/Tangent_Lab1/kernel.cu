#include "hip/hip_runtime.h"
﻿
#include "Utility.h"
#include "tangent.h"

using namespace std;

#define N 1000000
#define Block_Size 200

hipError_t tangentCuda(double* res, double* arr, int size)
{
	double* arr_device = 0;
	double* res_device = 0;

	hipError_t cudaStatus;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipSetDevice(0);

	hipMalloc((void**)&res_device, size * sizeof(double));
	hipMalloc((void**)&arr_device, size * sizeof(double));

	hipMemcpy(arr_device, arr, size * sizeof(double), hipMemcpyHostToDevice);

	hipEventRecord(start);
	tangentKernel << <N / Block_Size, Block_Size >> > (res_device, arr_device, size);
	hipEventRecord(stop);

	hipGetLastError();

	hipDeviceSynchronize();

	cudaStatus = hipMemcpy(res, res_device, size * sizeof(double), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);

	float time_elapsed = 0;
	hipEventElapsedTime(&time_elapsed, start, stop);

	printf("GPU done in %f milliseconds\n", time_elapsed);

	hipFree(res_device);
	hipFree(arr_device);

	return cudaStatus;
}

void tangentCpu(double* res, double* arr, int size)
{
	float start = ((float)clock() / (CLOCKS_PER_SEC)) * 1000.0;

	tangent(res, arr, N); // On CPU

	float end = ((float)clock() / (CLOCKS_PER_SEC)) * 1000.0;

	float time_elapsed = end - start;
	printf("CPU done in %f milliseconds\n", time_elapsed);
}

int main()
{
	double* arr = new double[N];
	double* res = new double[N];

	createArr(arr, N);

	cout << "Result: " << endl;

	hipError_t cudaStatus = tangentCuda(res, arr, N); // On GPU
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "tanWithCuda failed!");
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	tangentCpu(res, arr, N);

	return 0;
}